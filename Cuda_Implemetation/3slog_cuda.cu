#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

extern "C" {
#include "3slog_cuda.h"
}

__global__ void log_motion_estimation_cuda(uint8 *current, uint8 *previous, int *vectors_x, int *vectors_y,
                                           int *M_B, int *N_B, int *B, int *M, int *N) {
    //obtain idx;
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= ((*M_B) * (*N_B))) return;
    int x, y;

    x = id / (*M_B);
    y = id % (*M_B);


    int dd;
    for (dd = 4; dd > 1; dd--) {  //--> gives 4 3 2
        int step = 0;
        if (dd == 4) {
            //d=4;
            step = 4;
        } else if (dd == 3) {
            step = 2;
        } else if (dd == 2) {
            step = 1;
        } else {
            continue;
        }

        int min = 255 * (*B) * (*B);
        int bestx, besty, i, j, k, l;
        for (i = -step; i < step + 1; i += step)      /* For all candidate blocks */
            for (j = -step; j < step + 1; j += step) {
                int dist = 0;
                for (k = 0; k < (*B); k++)        /* For all pixels in the block */
                    for (l = 0; l < (*B); l++) {
                        int tmp9 = vectors_x[x * (*M_B) + y];
                        int p1, p2;
                        p1 = current[((*B) * x + k) * (*M) + (*B) * y + l];
                        if (((*B) * x + tmp9 + i + k) < 0 || ((*B) * x + tmp9 + i + k) > ((*N) - 1) ||
                            ((*B) * y + tmp9 + j + l) < 0 || ((*B) * y + tmp9 + j + l) > ((*M) - 1)) {
                            p2 = 0;
                        } else {
                            p2 = previous[((*B) * x + tmp9 + i + k) * (*M) + (*B) * y + tmp9 + j + l];
                        }

                        dist += abs(p1 - p2);
                    }
                if (dist < min) {
                    min = dist;
                    bestx = i;
                    besty = j;
                }
            }

        int at = x * (*M_B) + y;

        vectors_x[at] += bestx;

        vectors_y[at] += besty;

    }
}


__global__ void log_motion_estimation_cuda2(uint8 *current, uint8 *previous, int *vectors_x, int *vectors_y,
                                            int M_B, int N_B, int B, int M, int N) {
    //obtain id;
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    if (id >= (M_B * N_B)) {
        //printf("id=%d \t M_B*N_B=%d \t \n", id, M_B*N_B);
        return;
    }
    int x, y;
    x = id / M_B;
    y = id % M_B;
    vectors_x[id] = 0;
    vectors_y[id] = 0;
//    if(id==8023||id==8091||id==8159) {
//        printf("id=%d \t x=%d \t y=%d \n", id,x,y);
//    }
//
//    y=id;
//    x=0;
//    while(y>=M_B){
//        y=y-M_B;
//        x++;
//    }

    int dd;
    for (dd = 4; dd > 1; dd--) {  //--> gives 4 3 2
        int step = 0;
        if (dd == 4) {
            //d=4;
            step = 4;
        } else if (dd == 3) {
            step = 2;
        } else if (dd == 2) {
            step = 1;
        } else {
            continue;
        }

        int min = 255 * B * B;
        int bestx, besty, i, j, k, l;
        for (i = -step; i < step + 1; i += step)      /* For all candidate blocks */
            for (j = -step; j < step + 1; j += step) {
                int dist = 0;
                for (k = 0; k < B; k++)        /* For all pixels in the block */
                    for (l = 0; l < B; l++) {
                        int tmp9 = vectors_x[id];
                        int p1, p2;
                        p1 = current[(B * x + k) * M + B * y + l];
                        if ((B * x + tmp9 + i + k) < 0 || (B * x + tmp9 + i + k) > (N - 1) ||
                            (B * y + tmp9 + j + l) < 0 || (B * y + tmp9 + j + l) > (M - 1)) {
                            p2 = 0;
                        } else {
                            p2 = previous[(B * x + tmp9 + i + k) * M + B * y + tmp9 + j + l];
                        }

                        dist += abs(p1 - p2);
                    }
                if (dist < min) {
                    min = dist;
                    bestx = i;
                    besty = j;
                }
            }


        vectors_x[id] += bestx;

        vectors_y[id] += besty;

    }
}

extern "C"
void make_zeros(int *vectors_x, int *vectors_y, int N_B, int M_B) {
    for (int c = 0; c < N_B; c++)
        for (int b = 0; b < M_B; b++) {
            vectors_x[c * M_B + b] = 0;
            vectors_y[c * M_B + b] = 0;
        }
}

extern "C"
void initKernelAndStartIt(uint8 *current, uint8 *previous, int *vectors_x, int *vectors_y) {
    hipError_t err = hipSuccess;
    //make_zeros(vectors_x, vectors_y, t_N_B, t_M_B);

    if ((hipMemcpy(d_current, current, framesize, hipMemcpyHostToDevice) != hipSuccess) ||
        (hipMemcpy(d_previous, previous, framesize, hipMemcpyHostToDevice) != hipSuccess) ||
        (hipMemcpy(d_vectors_x, vectors_x, sizeof(int) * totalsize, hipMemcpyHostToDevice) != hipSuccess) ||
        (hipMemcpy(d_vectors_y, vectors_y, sizeof(int) * totalsize, hipMemcpyHostToDevice) != hipSuccess)) {
        err = hipGetLastError();
        if (err != hipSuccess) {
            printf("Failed to memcpy (error code: %s)!\n", hipGetErrorString(err));
        }
    }

    //log_motion_estimation_cuda <<< (int)ceil(totalsize / cudaThreads) , cudaThreads >>>(d_current, d_previous, d_vectors_x, d_vectors_y, d_M_B, d_N_B, d_B, d_M, d_N);
    log_motion_estimation_cuda2 <<< totalsize / cudaThreads + 1, cudaThreads >>>
                                                                  (d_current, d_previous, d_vectors_x, d_vectors_y, t_M_B, t_N_B, t_B, t_M, t_N);

    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Failed to launch add kernel (error code: %s)!\n", hipGetErrorString(err));
    }

    hipDeviceSynchronize();
    if ((hipMemcpy(vectors_x, d_vectors_x, sizeof(int) * totalsize, hipMemcpyDeviceToHost) != hipSuccess) ||
        (hipMemcpy(vectors_y, d_vectors_y, sizeof(int) * totalsize, hipMemcpyDeviceToHost) != hipSuccess)) {
        err = hipGetLastError();
        if (err != hipSuccess) {
            printf("Failed to memcpy (error code: %s)!\n", hipGetErrorString(err));
        }
    }

}



extern "C"
void initValuesAndAllocateMemory(int M_B, int N_B, int B, int M, int N) {
    totalsize = (N_B) * (M_B);
    t_N_B = N_B;
    t_M_B = M_B;
    t_B = B;
    t_M = M;
    t_N = N;
    framesize = sizeof(int) * (N + 1) * (M + 1);


    d_current = (uint8 *) fixed_cudaMalloc(framesize);
    d_previous = (uint8 *) fixed_cudaMalloc(framesize);
    d_vectors_y = (int *) fixed_cudaMalloc(totalsize * sizeof(int));
    d_vectors_x = (int *) fixed_cudaMalloc(totalsize * sizeof(int));

    if ((hipMalloc(&d_M_B, sizeof(int)) != hipSuccess) ||
        (hipMalloc(&d_N_B, sizeof(int)) != hipSuccess) ||
        (hipMalloc(&d_B, sizeof(int)) != hipSuccess) ||
        (hipMalloc(&d_M, sizeof(int)) != hipSuccess) ||
        (hipMalloc(&d_N, sizeof(int)) != hipSuccess)) {

    }

    if ((hipMemcpy(d_M_B, &M_B, sizeof(int), hipMemcpyHostToDevice) != hipSuccess) ||
        (hipMemcpy(d_N_B, &N_B, sizeof(int), hipMemcpyHostToDevice) != hipSuccess) ||
        (hipMemcpy(d_B, &B, sizeof(int), hipMemcpyHostToDevice) != hipSuccess) ||
        (hipMemcpy(d_M, &M, sizeof(int), hipMemcpyHostToDevice) != hipSuccess) ||
        (hipMemcpy(d_N, &N, sizeof(int), hipMemcpyHostToDevice) != hipSuccess)) {

    }


}
extern "C"
void freeMemory() {
    hipFree(d_vectors_y);
    hipFree(d_vectors_x);
    hipFree(d_current);
    hipFree(d_previous);
    hipFree(d_M);
    hipFree(d_N);
    hipFree(d_B);
    hipFree(d_M_B);
    hipFree(d_N_B);

}

extern "C"
void *fixed_cudaMalloc(size_t len) {
    void *p;
    if (hipMalloc(&p, len) == hipSuccess) return p;
    return 0;
}
